#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include "checks.cuh"
#include "spherical_harmonics.cuh"

template <typename T, unsigned int N_SH>
__global__ void precompute_rgb_from_sh_kernel(
    const T* __restrict__ xyz,
    const T* __restrict__ sh_coeff,
    const T* __restrict__ camera_T_world,
    const unsigned int N,
    T* __restrict__ rgb
) {
    const int gaussian_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (gaussian_idx >= N) {
        return;
    }

    if (N_SH == 1) {
        #pragma unroll
        for (int channel = 0; channel < 3; channel++) {
            rgb[gaussian_idx * 3 + channel] = sh_coeff[gaussian_idx * 3 + channel];
        }
    } else {
        // compute normalized view direction
        const T camera_center[3] = {
            camera_T_world[0 * 4 + 3], camera_T_world[1 * 4 + 3], camera_T_world[2 * 4 + 3]};
        T view_dir[3] = {
            xyz[gaussian_idx * 3 + 0] - camera_center[0],
            xyz[gaussian_idx * 3 + 1] - camera_center[1],
            xyz[gaussian_idx * 3 + 2] - camera_center[2]};
        const T r_view_dir_norm = rsqrt(
            view_dir[0] * view_dir[0] + view_dir[1] * view_dir[1] + view_dir[2] * view_dir[2]
        );
        #pragma unroll
        for (int i = 0; i < 3; ++i) {
            view_dir[i] *= r_view_dir_norm;
        }

        T sh_at_view_dir[N_SH];
        compute_sh_coeffs_for_view_dir<T, N_SH>(view_dir, sh_at_view_dir);

        #pragma unroll
        for (int channel = 0; channel < 3; channel++) {
            T temp_rgb = 0.0;
            #pragma unroll
            for (int sh_idx = 0; sh_idx < N_SH; sh_idx++) {
                temp_rgb += sh_at_view_dir[sh_idx] *
                            sh_coeff[gaussian_idx * N_SH * 3 + N_SH * channel + sh_idx];
            }
            // divide by SH_0 to maintain compatibility with downstream rasterizer
            temp_rgb *= r_SH_0;
            // set value on output
            rgb[gaussian_idx * 3 + channel] = temp_rgb;
        }
    }
}

template <typename T, unsigned int N_SH>
__global__ void precompute_rgb_from_sh_backward_kernel(
    const T* __restrict__ xyz,
    const T* __restrict__ camera_T_world,
    const T* __restrict__ grad_rgb,
    const unsigned int N,
    T* __restrict__ grad_sh
) {
    const int gaussian_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (gaussian_idx >= N) {
        return;
    }
    if (N_SH == 1) {
        #pragma unroll
        for (int channel = 0; channel < 3; channel++) {
            grad_sh[gaussian_idx * 3 + channel] = grad_rgb[gaussian_idx * 3 + channel];
        }
    } else {
        // compute normalized view direction
        const T camera_center[3] = {
            camera_T_world[0 * 4 + 3], camera_T_world[1 * 4 + 3], camera_T_world[2 * 4 + 3]};
        T view_dir[3] = {
            xyz[gaussian_idx * 3 + 0] - camera_center[0],
            xyz[gaussian_idx * 3 + 1] - camera_center[1],
            xyz[gaussian_idx * 3 + 2] - camera_center[2]};
        const T r_view_dir_norm = rsqrt(
            view_dir[0] * view_dir[0] + view_dir[1] * view_dir[1] + view_dir[2] * view_dir[2]
        );
        #pragma unroll
        for (int i = 0; i < 3; ++i) {
            view_dir[i] *= r_view_dir_norm;
        }

        T sh_at_view_dir[N_SH];
        compute_sh_coeffs_for_view_dir<T, N_SH>(view_dir, sh_at_view_dir);

        // make local copy and undo scaling by SH_0
        T grad_rgb_local[3] = {
            grad_rgb[gaussian_idx * 3 + 0] * r_SH_0,
            grad_rgb[gaussian_idx * 3 + 1] * r_SH_0,
            grad_rgb[gaussian_idx * 3 + 2] * r_SH_0};

        #pragma unroll
        for (int channel = 0; channel < 3; channel++) {
            #pragma unroll
            for (int sh_idx = 0; sh_idx < N_SH; sh_idx++) {
                grad_sh[gaussian_idx * N_SH * 3 + N_SH * channel + sh_idx] =
                    grad_rgb_local[channel] * sh_at_view_dir[sh_idx];
            }
        }
    }
}

void precompute_rgb_from_sh_cuda(
    const torch::Tensor xyz,
    const torch::Tensor sh_coeff,
    const torch::Tensor camera_T_world,
    torch::Tensor rgb
) {
    CHECK_VALID_INPUT(xyz);
    CHECK_VALID_INPUT(sh_coeff);
    CHECK_VALID_INPUT(camera_T_world);
    CHECK_VALID_INPUT(rgb);

    const int N = xyz.size(0);
    int num_sh_coeff;
    if (sh_coeff.dim() == 3) {
        num_sh_coeff = sh_coeff.size(2);
    } else {
        num_sh_coeff = 1;
    }

    const int max_threads_per_block = 1024;
    const int num_blocks = (N + max_threads_per_block - 1) / max_threads_per_block;
    dim3 gridsize(num_blocks, 1, 1);
    dim3 blocksize(max_threads_per_block, 1, 1);

    if (xyz.dtype() == torch::kFloat32) {
        if (num_sh_coeff == 1) {
            precompute_rgb_from_sh_kernel<float, 1><<<gridsize, blocksize>>>(
                xyz.data_ptr<float>(),
                sh_coeff.data_ptr<float>(),
                camera_T_world.data_ptr<float>(),
                N,
                rgb.data_ptr<float>()
            );
        } else if (num_sh_coeff == 4) {
            precompute_rgb_from_sh_kernel<float, 4><<<gridsize, blocksize>>>(
                xyz.data_ptr<float>(),
                sh_coeff.data_ptr<float>(),
                camera_T_world.data_ptr<float>(),
                N,
                rgb.data_ptr<float>()
            );
        } else if (num_sh_coeff == 9) {
            precompute_rgb_from_sh_kernel<float, 9><<<gridsize, blocksize>>>(
                xyz.data_ptr<float>(),
                sh_coeff.data_ptr<float>(),
                camera_T_world.data_ptr<float>(),
                N,
                rgb.data_ptr<float>()
            );
        } else if (num_sh_coeff == 16) {
            precompute_rgb_from_sh_kernel<float, 16><<<gridsize, blocksize>>>(
                xyz.data_ptr<float>(),
                sh_coeff.data_ptr<float>(),
                camera_T_world.data_ptr<float>(),
                N,
                rgb.data_ptr<float>()
            );
        } else {
            AT_ERROR("Unsupported number of SH coefficients: ", num_sh_coeff);
        }
    } else if (xyz.dtype() == torch::kFloat64) {
        if (num_sh_coeff == 1) {
            precompute_rgb_from_sh_kernel<double, 1><<<gridsize, blocksize>>>(
                xyz.data_ptr<double>(),
                sh_coeff.data_ptr<double>(),
                camera_T_world.data_ptr<double>(),
                N,
                rgb.data_ptr<double>()
            );
        } else if (num_sh_coeff == 4) {
            precompute_rgb_from_sh_kernel<double, 4><<<gridsize, blocksize>>>(
                xyz.data_ptr<double>(),
                sh_coeff.data_ptr<double>(),
                camera_T_world.data_ptr<double>(),
                N,
                rgb.data_ptr<double>()
            );
        } else if (num_sh_coeff == 9) {
            precompute_rgb_from_sh_kernel<double, 9><<<gridsize, blocksize>>>(
                xyz.data_ptr<double>(),
                sh_coeff.data_ptr<double>(),
                camera_T_world.data_ptr<double>(),
                N,
                rgb.data_ptr<double>()
            );
        } else if (num_sh_coeff == 16) {
            precompute_rgb_from_sh_kernel<double, 16><<<gridsize, blocksize>>>(
                xyz.data_ptr<double>(),
                sh_coeff.data_ptr<double>(),
                camera_T_world.data_ptr<double>(),
                N,
                rgb.data_ptr<double>()
            );
        } else {
            AT_ERROR("Unsupported number of SH coefficients: ", num_sh_coeff);
        }
    } else {
        AT_ERROR("Unsupported data type: ", xyz.dtype());
    }
    hipDeviceSynchronize();
}

void precompute_rgb_from_sh_backward_cuda(
    const torch::Tensor xyz,
    const torch::Tensor camera_T_world,
    const torch::Tensor grad_rgb,
    torch::Tensor grad_sh
) {
    CHECK_VALID_INPUT(xyz);
    CHECK_VALID_INPUT(camera_T_world);
    CHECK_VALID_INPUT(grad_rgb);
    CHECK_VALID_INPUT(grad_sh);

    const int N = xyz.size(0);
    int num_sh_coeff;
    if (grad_sh.dim() == 3) {
        num_sh_coeff = grad_sh.size(2);
    } else {
        num_sh_coeff = 1;
    }

    const int max_threads_per_block = 1024;
    const int num_blocks = (N + max_threads_per_block - 1) / max_threads_per_block;
    dim3 gridsize(num_blocks, 1, 1);
    dim3 blocksize(max_threads_per_block, 1, 1);

    if (xyz.dtype() == torch::kFloat32) {
        if (num_sh_coeff == 1) {
            precompute_rgb_from_sh_backward_kernel<float, 1><<<gridsize, blocksize>>>(
                xyz.data_ptr<float>(),
                camera_T_world.data_ptr<float>(),
                grad_rgb.data_ptr<float>(),
                N,
                grad_sh.data_ptr<float>()
            );
        } else if (num_sh_coeff == 4) {
            precompute_rgb_from_sh_backward_kernel<float, 4><<<gridsize, blocksize>>>(
                xyz.data_ptr<float>(),
                camera_T_world.data_ptr<float>(),
                grad_rgb.data_ptr<float>(),
                N,
                grad_sh.data_ptr<float>()
            );
        } else if (num_sh_coeff == 9) {
            precompute_rgb_from_sh_backward_kernel<float, 9><<<gridsize, blocksize>>>(
                xyz.data_ptr<float>(),
                camera_T_world.data_ptr<float>(),
                grad_rgb.data_ptr<float>(),
                N,
                grad_sh.data_ptr<float>()
            );
        } else if (num_sh_coeff == 16) {
            precompute_rgb_from_sh_backward_kernel<float, 16><<<gridsize, blocksize>>>(
                xyz.data_ptr<float>(),
                camera_T_world.data_ptr<float>(),
                grad_rgb.data_ptr<float>(),
                N,
                grad_sh.data_ptr<float>()
            );
        } else {
            AT_ERROR("Unsupported number of SH coefficients: ", num_sh_coeff);
        }
    } else if (xyz.dtype() == torch::kFloat64) {
        if (num_sh_coeff == 1) {
            precompute_rgb_from_sh_backward_kernel<double, 1><<<gridsize, blocksize>>>(
                xyz.data_ptr<double>(),
                camera_T_world.data_ptr<double>(),
                grad_rgb.data_ptr<double>(),
                N,
                grad_sh.data_ptr<double>()
            );
        } else if (num_sh_coeff == 4) {
            precompute_rgb_from_sh_backward_kernel<double, 4><<<gridsize, blocksize>>>(
                xyz.data_ptr<double>(),
                camera_T_world.data_ptr<double>(),
                grad_rgb.data_ptr<double>(),
                N,
                grad_sh.data_ptr<double>()
            );
        } else if (num_sh_coeff == 9) {
            precompute_rgb_from_sh_backward_kernel<double, 9><<<gridsize, blocksize>>>(
                xyz.data_ptr<double>(),
                camera_T_world.data_ptr<double>(),
                grad_rgb.data_ptr<double>(),
                N,
                grad_sh.data_ptr<double>()
            );
        } else if (num_sh_coeff == 16) {
            precompute_rgb_from_sh_backward_kernel<double, 16><<<gridsize, blocksize>>>(
                xyz.data_ptr<double>(),
                camera_T_world.data_ptr<double>(),
                grad_rgb.data_ptr<double>(),
                N,
                grad_sh.data_ptr<double>()
            );
        } else {
            AT_ERROR("Unsupported number of SH coefficients: ", num_sh_coeff);
        }
    } else {
        AT_ERROR("Unsupported data type: ", xyz.dtype());
    }
    hipDeviceSynchronize();
}
