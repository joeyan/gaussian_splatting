#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "checks.cuh"
#include "matrix.cuh"

template <typename T>
__global__ void
camera_projection_kernel(const T* __restrict__ xyz, const T* __restrict__ K, const int N, T* uv) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) {
        return;
    }
    // u = fx * X / Z + cx
    uv[i * 2 + 0] = K[0] * xyz[i * 3 + 0] / xyz[i * 3 + 2] + K[2];
    // v = fy * Y / Z + cy
    uv[i * 2 + 1] = K[4] * xyz[i * 3 + 1] / xyz[i * 3 + 2] + K[5];
}

void camera_projection_cuda(torch::Tensor xyz, torch::Tensor K, torch::Tensor uv) {
    CHECK_VALID_INPUT(xyz);
    CHECK_VALID_INPUT(K);
    CHECK_VALID_INPUT(uv);

    const int N = xyz.size(0);
    TORCH_CHECK(xyz.size(1) == 3, "xyz must have shape Nx3");
    TORCH_CHECK(K.size(0) == 3, "K must have shape 3x3");
    TORCH_CHECK(K.size(1) == 3, "K must have shape 3x3");
    TORCH_CHECK(uv.size(0) == N, "uv must have shape Nx2");
    TORCH_CHECK(uv.size(1) == 2, "uv must have shape Nx2");

    const int max_threads_per_block = 1024;
    const int num_blocks = (N + max_threads_per_block - 1) / max_threads_per_block;
    dim3 gridsize(num_blocks, 1, 1);
    dim3 blocksize(max_threads_per_block, 1, 1);

    if (xyz.dtype() == torch::kFloat32) {
        CHECK_FLOAT_TENSOR(K);
        CHECK_FLOAT_TENSOR(uv);
        camera_projection_kernel<float><<<gridsize, blocksize>>>(
            xyz.data_ptr<float>(), K.data_ptr<float>(), N, uv.data_ptr<float>()
        );
    } else if (xyz.dtype() == torch::kFloat64) {
        CHECK_DOUBLE_TENSOR(K);
        CHECK_DOUBLE_TENSOR(uv);
        camera_projection_kernel<double><<<gridsize, blocksize>>>(
            xyz.data_ptr<double>(), K.data_ptr<double>(), N, uv.data_ptr<double>()
        );
    } else {
        AT_ERROR("Inputs must be float32 or float64");
    }
    hipDeviceSynchronize();
}

template <typename T>
__global__ void compute_sigma_world_kernel(
    const T* __restrict__ quaternion,
    const T* __restrict__ scale,
    const int N,
    T* sigma_world
) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) {
        return;
    }
    T qw = quaternion[i * 4 + 0];
    T qx = quaternion[i * 4 + 1];
    T qy = quaternion[i * 4 + 2];
    T qz = quaternion[i * 4 + 3];

    T norm = sqrt(qx * qx + qy * qy + qz * qz + qw * qw);

    // // zero magnitude quaternion is not valid
    qx /= norm;
    qy /= norm;
    qz /= norm;
    qw /= norm;

    T r00 = 1 - 2 * qy * qy - 2 * qz * qz;
    T r01 = 2 * qx * qy - 2 * qz * qw;
    T r02 = 2 * qx * qz + 2 * qy * qw;
    T r10 = 2 * qx * qy + 2 * qz * qw;
    T r11 = 1 - 2 * qx * qx - 2 * qz * qz;
    T r12 = 2 * qy * qz - 2 * qx * qw;
    T r20 = 2 * qx * qz - 2 * qy * qw;
    T r21 = 2 * qy * qz + 2 * qx * qw;
    T r22 = 1 - 2 * qx * qx - 2 * qy * qy;

    T sx = exp(scale[i * 3 + 0]);
    T sy = exp(scale[i * 3 + 1]);
    T sz = exp(scale[i * 3 + 2]);

    T sx_sq = sx * sx;
    T sy_sq = sy * sy;
    T sz_sq = sz * sz;

    sigma_world[i * 9 + 0] = r00 * r00 * sx_sq + r01 * r01 * sy_sq + r02 * r02 * sz_sq;
    sigma_world[i * 9 + 1] = r00 * r10 * sx_sq + r01 * r11 * sy_sq + r02 * r12 * sz_sq;
    sigma_world[i * 9 + 2] = r00 * r20 * sx_sq + r01 * r21 * sy_sq + r02 * r22 * sz_sq;

    sigma_world[i * 9 + 3] = r00 * r10 * sx_sq + r01 * r11 * sy_sq + r02 * r12 * sz_sq;
    sigma_world[i * 9 + 4] = r10 * r10 * sx_sq + r11 * r11 * sy_sq + r12 * r12 * sz_sq;
    sigma_world[i * 9 + 5] = r10 * r20 * sx_sq + r11 * r21 * sy_sq + r12 * r22 * sz_sq;

    sigma_world[i * 9 + 6] = r00 * r20 * sx_sq + r01 * r21 * sy_sq + r02 * r22 * sz_sq;
    sigma_world[i * 9 + 7] = r10 * r20 * sx_sq + r11 * r21 * sy_sq + r12 * r22 * sz_sq;
    sigma_world[i * 9 + 8] = r20 * r20 * sx_sq + r21 * r21 * sy_sq + r22 * r22 * sz_sq;
}

void compute_sigma_world_cuda(
    torch::Tensor quaternion,
    torch::Tensor scale,
    torch::Tensor sigma_world
) {
    CHECK_VALID_INPUT(quaternion);
    CHECK_VALID_INPUT(scale);
    CHECK_VALID_INPUT(sigma_world);

    const int N = quaternion.size(0);
    TORCH_CHECK(quaternion.size(1) == 4, "quaternion must have shape Nx4");
    TORCH_CHECK(scale.size(0) == N, "scale must have shape Nx1");
    TORCH_CHECK(sigma_world.size(0) == N, "sigma_world must have shape Nx3x3");
    TORCH_CHECK(sigma_world.size(1) == 3, "sigma_world must have shape Nx3x3");
    TORCH_CHECK(sigma_world.size(2) == 3, "sigma_world must have shape Nx3x3");

    // can probably update this to improve perf
    const int max_threads_per_block = 1024;
    const int num_blocks = (N + max_threads_per_block - 1) / max_threads_per_block;
    dim3 gridsize(num_blocks, 1, 1);
    dim3 blocksize(max_threads_per_block, 1, 1);

    if (quaternion.dtype() == torch::kFloat32) {
        CHECK_FLOAT_TENSOR(scale);
        CHECK_FLOAT_TENSOR(sigma_world);
        compute_sigma_world_kernel<float><<<gridsize, blocksize>>>(
            quaternion.data_ptr<float>(), scale.data_ptr<float>(), N, sigma_world.data_ptr<float>()
        );
    } else if (quaternion.dtype() == torch::kFloat64) {
        CHECK_DOUBLE_TENSOR(scale);
        CHECK_DOUBLE_TENSOR(sigma_world);
        compute_sigma_world_kernel<double><<<gridsize, blocksize>>>(
            quaternion.data_ptr<double>(),
            scale.data_ptr<double>(),
            N,
            sigma_world.data_ptr<double>()
        );
    } else {
        AT_ERROR("Inputs must be float32 or float64");
    }
    hipDeviceSynchronize();
}

template <typename T>
__global__ void compute_projection_jacobian_kernel(
    const T* __restrict__ xyz,
    const T* __restrict__ K,
    const int N,
    T* J
) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) {
        return;
    }
    T x = xyz[i * 3 + 0];
    T y = xyz[i * 3 + 1];
    T z = xyz[i * 3 + 2];

    J[i * 6 + 0] = K[0] / z;
    J[i * 6 + 1] = 0;
    J[i * 6 + 2] = -K[0] * x / (z * z);
    J[i * 6 + 3] = 0;
    J[i * 6 + 4] = K[4] / z;
    J[i * 6 + 5] = -K[4] * y / (z * z);
}

void compute_projection_jacobian_cuda(torch::Tensor xyz, torch::Tensor K, torch::Tensor J) {
    CHECK_VALID_INPUT(xyz);
    CHECK_VALID_INPUT(K);
    CHECK_VALID_INPUT(J);

    const int N = xyz.size(0);
    TORCH_CHECK(xyz.size(1) == 3, "xyz must have shape Nx3");
    TORCH_CHECK(K.size(0) == 3, "K must have shape 3x3");
    TORCH_CHECK(K.size(1) == 3, "K must have shape 3x3");
    TORCH_CHECK(J.size(0) == N, "J must have shape Nx2x3");
    TORCH_CHECK(J.size(1) == 2, "J must have shape Nx2x3");
    TORCH_CHECK(J.size(2) == 3, "J must have shape Nx2x3");

    const int max_threads_per_block = 1024;
    const int num_blocks = (N + max_threads_per_block - 1) / max_threads_per_block;
    dim3 gridsize(num_blocks, 1, 1);
    dim3 blocksize(max_threads_per_block, 1, 1);

    if (xyz.dtype() == torch::kFloat32) {
        CHECK_FLOAT_TENSOR(K);
        CHECK_FLOAT_TENSOR(J);

        compute_projection_jacobian_kernel<float><<<gridsize, blocksize>>>(
            xyz.data_ptr<float>(), K.data_ptr<float>(), N, J.data_ptr<float>()
        );
    } else if (xyz.dtype() == torch::kFloat64) {
        CHECK_DOUBLE_TENSOR(K);
        CHECK_DOUBLE_TENSOR(J);
        compute_projection_jacobian_kernel<double><<<gridsize, blocksize>>>(
            xyz.data_ptr<double>(), K.data_ptr<double>(), N, J.data_ptr<double>()
        );
    } else {
        AT_ERROR("Inputs must be float32 or float64");
    }
}

template <typename T>
__global__ void compute_conic_kernel(
    const T* __restrict__ sigma_world,
    const T* __restrict__ J,
    const T* __restrict__ world_T_image,
    const int N,
    T* conic
) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) {
        return;
    }
    // get rotation matrix
    T W[9];
    W[0] = world_T_image[0];
    W[1] = world_T_image[1];
    W[2] = world_T_image[2];
    W[3] = world_T_image[4];
    W[4] = world_T_image[5];
    W[5] = world_T_image[6];
    W[6] = world_T_image[8];
    W[7] = world_T_image[9];
    W[8] = world_T_image[10];

    // compute JW = J * W)
    T JW[6];
    matrix_multiply<T>(J + i * 6, W, JW, 2, 3, 3);

    // compute JWSigma = JW * sigma_world
    T JWSigma[6];
    matrix_multiply<T>(JW, sigma_world + i * 9, JWSigma, 2, 3, 3);

    T JW_t[6];
    transpose<T>(JW, JW_t, 2, 3);

    // compute sigma_image = JWSigma @ JW_t
    T sigma_image[4];
    matrix_multiply<T>(JWSigma, JW_t, sigma_image, 2, 3, 2);

    // write to conic
    conic[i * 3 + 0] = sigma_image[0];
    // they are also equal but this keeps the pytorch autograd check happy
    conic[i * 3 + 1] = sigma_image[1] + sigma_image[2];
    conic[i * 3 + 2] = sigma_image[3];
}

void compute_conic_cuda(
    torch::Tensor sigma_world,
    torch::Tensor J,
    torch::Tensor world_T_image,
    torch::Tensor conic
) {
    CHECK_VALID_INPUT(sigma_world);
    CHECK_VALID_INPUT(J);
    CHECK_VALID_INPUT(world_T_image);
    CHECK_VALID_INPUT(conic);

    const int N = sigma_world.size(0);
    TORCH_CHECK(sigma_world.size(1) == 3, "sigma_world must have shape Nx3x3");
    TORCH_CHECK(sigma_world.size(2) == 3, "sigma_world must have shape Nx3x3");
    TORCH_CHECK(J.size(0) == N, "J must have shape Nx2x3");
    TORCH_CHECK(J.size(1) == 2, "J must have shape Nx2x3");
    TORCH_CHECK(J.size(2) == 3, "J must have shape Nx2x3");
    TORCH_CHECK(world_T_image.size(0) == 4, "world_T_image must have shape 4x4");
    TORCH_CHECK(world_T_image.size(1) == 4, "world_T_image must have shape 4x4");
    TORCH_CHECK(conic.size(0) == N, "conic must have shape Nx3");
    TORCH_CHECK(conic.size(1) == 3, "conic must have shape Nx3");

    const int max_threads_per_block = 1024;
    const int num_blocks = (N + max_threads_per_block - 1) / max_threads_per_block;
    dim3 gridsize(num_blocks, 1, 1);
    dim3 blocksize(max_threads_per_block, 1, 1);

    if (sigma_world.dtype() == torch::kFloat32) {
        CHECK_FLOAT_TENSOR(J);
        CHECK_FLOAT_TENSOR(world_T_image);
        CHECK_FLOAT_TENSOR(conic);
        compute_conic_kernel<float><<<gridsize, blocksize>>>(
            sigma_world.data_ptr<float>(),
            J.data_ptr<float>(),
            world_T_image.data_ptr<float>(),
            N,
            conic.data_ptr<float>()
        );
    } else if (sigma_world.dtype() == torch::kFloat64) {
        CHECK_DOUBLE_TENSOR(J);
        CHECK_DOUBLE_TENSOR(world_T_image);
        CHECK_DOUBLE_TENSOR(conic);
        compute_conic_kernel<double><<<gridsize, blocksize>>>(
            sigma_world.data_ptr<double>(),
            J.data_ptr<double>(),
            world_T_image.data_ptr<double>(),
            N,
            conic.data_ptr<double>()
        );
    } else {
        AT_ERROR("Inputs must be float32 or float64");
    }
}
